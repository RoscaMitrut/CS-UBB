#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

const char* MATRIX_A_FILE = "matrixA.txt";
const char* MATRIX_B_FILE = "matrixB.txt";
const char* OUTPUT_FILE = "output.txt";
const char* LOG_FILE = "log.csv";

void readMatrixFromFile(const char* filename, int** matrix, int* rows, int* cols) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    fscanf(file, "%d %d", rows, cols);
    *matrix = (int*)malloc((*rows) * (*cols) * sizeof(int));

    for (int i = 0; i < (*rows) * (*cols); i++) {
        fscanf(file, "%d", &(*matrix)[i]);
    }

    fclose(file);
}

void writeMatrixToFile(const char* filename, const int* matrix, int rows, int cols) {
    FILE* file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    fprintf(file, "%d %d\n", rows, cols);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%d ", matrix[i * cols + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

void logExecutionTime(const char* logFile, int rowsA, int colsA, int rowsB, int colsB, double timeTaken) {
    FILE* file = fopen(logFile, "a");
    if (!file) {
        fprintf(stderr, "Failed to open log file %s\n", logFile);
        exit(EXIT_FAILURE);
    }
    fprintf(file, "%d,%d,%d,%d,%.6f\n", rowsA, colsA, rowsB, colsB, timeTaken);
    fclose(file);
}

__global__ void matrixMultiplyKernel(int* c, const int* a, const int* b, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index of C
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index of C

    if (row < rowsA && col < colsB) {
        int value = 0;
        for (int k = 0; k < colsA; k++) {
            value += a[row * colsA + k] * b[k * colsB + col];
        }
        c[row * colsB + col] = value;
    }
}

hipError_t matrixMultiplyWithCuda(int* c, const int* a, const int* b, int rowsA, int colsA, int colsB) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

    // Allocate GPU buffers for matrices.
    cudaStatus = hipMalloc((void**)&dev_a, rowsA * colsA * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for A!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, colsA * colsB * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for B!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, rowsA * colsB * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for C!\n");
        goto Error;
    }

    // Copy input matrices from to device.
    cudaStatus = hipMemcpy(dev_a, a, rowsA * colsA * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for A!\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, colsA * colsB * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for B!\n");
        goto Error;
    }

    // Define the block and grid dimensions.
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + threadsPerBlock.x - 1) / threadsPerBlock.x, (rowsA + threadsPerBlock.y - 1) / threadsPerBlock.y);


    matrixMultiplyKernel <<<blocksPerGrid, threadsPerBlock >>> (dev_c, dev_a, dev_b, rowsA, colsA, colsB);


    // Check for kernel launch errors.
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixMultiplyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Synchronize and check for errors.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
        goto Error;
    }

    // Copy the result matrix to host.
    cudaStatus = hipMemcpy(c, dev_c, rowsA * colsB * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for C!\n");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}

int main() {
    clock_t start = clock();

    int* a = NULL, * b = NULL, * c = NULL;
    int rowsA, colsA, rowsB, colsB;

    readMatrixFromFile(MATRIX_A_FILE, &a, &rowsA, &colsA);
    readMatrixFromFile(MATRIX_B_FILE, &b, &rowsB, &colsB);

    // Check if multiplication is possible
    if (colsA != rowsB) {
        fprintf(stderr, "Matrix multiplication not possible: colsA (%d) != rowsB (%d)\n", colsA, rowsB);
        free(a);
        free(b);
        return 1;
    }

    int rowsC = rowsA, colsC = colsB;
    c = (int*)malloc(rowsC * colsC * sizeof(int));


    hipError_t cudaStatus = matrixMultiplyWithCuda(c, a, b, rowsA, colsA, colsB);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixMultiplyWithCuda failed!\n");
        free(a);
        free(b);
        free(c);
        return 1;
    }

    writeMatrixToFile(OUTPUT_FILE, c, rowsC, colsC);

    clock_t end = clock();
    double timeTaken = ((double)(end - start)) / CLOCKS_PER_SEC;

    logExecutionTime(LOG_FILE, rowsA, colsA, rowsB, colsB, timeTaken);

    free(a);
    free(b);
    free(c);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }

    return 0;
}

